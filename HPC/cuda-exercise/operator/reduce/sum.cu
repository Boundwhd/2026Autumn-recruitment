#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 1024
#define BLOCK_SIZE 32


__global__ void vector_sum(float *A, float *B, int size) {
    __shared__ float sm[32];
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int warpID = threadIdx.x / warpSize;
    int laneID = threadIdx.x % warpSize;

    float val = (idx < size) ? A[idx] : 0.0f;
    
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }

    if (laneID == 0) {
        sm[warpID] = val;
    }
    __syncthreads();

    if (warpID == 0) {
        int warpNum = blockDim.x / warpSize;
        val = (laneID < warpNum) ? sm[laneID] : 0;
        for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }
        if(laneID == 0) atomicAdd(B, val);
    }
}

int main() {
    // 初始化数据
    int size = N * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(sizeof(float));  // 存储最终的求和结果
    h_B[0] = 0.0f;  // 初始化B为0

    // 填充输入向量A
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;  // 假设A的每个元素为1.0
    }

    // 分配设备内存
    float *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, sizeof(float));

    // 将数据从主机拷贝到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float), hipMemcpyHostToDevice);

    // 计算所需的线程块数量
    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    // 调用CUDA内核
    vector_sum<<<gridSize, blockSize>>>(d_A, d_B, N);

    // 检查内核执行是否成功
    hipDeviceSynchronize();

    // 将结果从设备复制回主机
    hipMemcpy(h_B, d_B, sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果
    printf("The sum of the vector is: %f\n", h_B[0]);

    // 清理内存
    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}