#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define N 1024  // A和C的行数
#define M 1024  // B的列数
#define K 1024  // A和B的列数（A的列数等于B的行数）

template<const int BLOCKSIZE>
__global__ void shared_mem_sgemm(float *A, float *B, float *C, int m, int n, int k) {
    const uint cROW = blockIdx.x;
    const uint cCol = blockIdx.y;

    __shared__ float AS[BLOCKSIZE * BLOCKSIZE];
    __shared__ float BS[BLOCKSIZE * BLOCKSIZE];

    const uint threadCol = threadIdx.x % BLOCKSIZE;
    const uint threadRow = threadIdx.x / BLOCKSIZE;

    A += cROW * K * BLOCKSIZE;
    B += cCol * BLOCKSIZE;
    C += cROW * N * BLOCKSIZE + cCol * BLOCKSIZE;

    float temp = 0.0f;
    for (int bkidx = 0; bkidx < k; bkidx += BLOCKSIZE) {
        AS[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
        BS[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];
        __syncthreads();

        A += BLOCKSIZE;
        B += BLOCKSIZE * n;

        for (int dotidx = 0; dotidx < BLOCKSIZE; dotidx++) {
            temp += AS[threadRow * BLOCKSIZE + dotidx] * BS[dotidx * BLOCKSIZE + threadCol];
        }

        __syncthreads();
    }
    C[threadRow * N + threadCol] = temp;
}

// CUDA主函数，调用shared_mem_sgemm内核
int main() {
    // 定义矩阵的大小
    int sizeA = M * K * sizeof(float);  // A矩阵的大小
    int sizeB = K * N * sizeof(float);  // B矩阵的大小
    int sizeC = M * N * sizeof(float);  // C矩阵的大小

    // 分配主机内存
    float *h_A = (float *)malloc(sizeA);
    float *h_B = (float *)malloc(sizeB);
    float *h_C = (float *)malloc(sizeC);

    // 初始化矩阵A和B
    for (int i = 0; i < M * K; i++) {
        h_A[i] = 1.0f;  // 设置A矩阵的每个元素为1.0
    }
    for (int i = 0; i < K * N; i++) {
        h_B[i] = 1.0f;  // 设置B矩阵的每个元素为1.0
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    // 将数据从主机拷贝到设备
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeC, hipMemcpyHostToDevice);

    // 定义线程块和网格的大小
    const int BLOCKSIZE = 32;  // 假设每个块有16x16线程
    dim3 threadsPerBlock(32 * 32);
    dim3 numBlocks((M + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);

    // 启动CUDA内核
    shared_mem_sgemm<BLOCKSIZE><<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);

    // 检查内核执行是否成功
    hipDeviceSynchronize();

    // 从设备拷贝结果到主机
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // 打印部分结果（这里只打印第一个元素）
    printf("C[0][0] = %f\n", h_C[0]);

    // 清理内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
