#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define N 1024  
#define M 1024  
#define K 1024  

template<const int BLOCKSIZE>
__global__ void global_mem_clo(float *A, float *B, float *C, int m, int n, int k){
    const uint cRow = blockidx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const uint cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    if (cRow < m && cCol < n) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[cRow * k + l] * B[l * N + cCol];
        }
        c[cRow * N + cCol] = sum;
    }
}