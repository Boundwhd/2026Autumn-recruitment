#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

void cudaCheck(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void init_matrix(float* add, size_t size, int value) {
    for (size_t i = 0; i < size; i++) {
        add[i] = value;
    }
}

// input (B, T, 3C)
// preatt att are (B, NH, T, T)
// output (B, T, C)
template<const int BLOCKSIZE>

__global__ void attention_query_key_kernel1(float* preatt, const float* inp, int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx;
    int total_threads = B * NH * T * T;

    if (idx < total_threads) {
        int t2 = idx % T;
        int t = (idx / T) % T;
        if (t2 > t) {
            preatt[idx] = -INFINITY;
            return;
        }
        int h = (idx / (T * T)) % NH;
        int b = idx / (NH * T * T);







    }
}

int main() {
    int B = 2;      // Btach-size
    int H = 8;      // Multi-Attention Head
    int T = 512;    // Q,K,V 矩阵行数量
    int C = 1024;   // Q,K,V 矩阵列数量

    float *h_in;
    float *d_in;
    float *d_out;
    float *out_cpu, *out_cuda;

    // 主机分配内存
    h_in = (float*)malloc(B * T * 3*C * sizeof(float));
    // Cuda分配内存
    hipMalloc((void**)&d_in, B * H * T * T * sizeof(float));

    int total_threads = B * H * T * T;
    int block_size = 1024;
    int grid_size = (total_threads + block_size - 1) / block_size; 

    
}