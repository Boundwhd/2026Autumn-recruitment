#include "gemm.cuh"
#include "hipblas.h"

void gemm_cublas(const float* A, const float* B, float* C, int M, int K, int N) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasHandle_t handle = nullptr;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    static float alpha = 1.0;
    static float beta = 0.0;

    hipEventRecord(start);
    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                N, M, K,
                &alpha,
                B, N,
                A, K,
                &beta,
                C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float GFLOPS_S = (2.0 * K * N * M) / milliseconds / 1e6;

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_cublas_v2: " << milliseconds << " ms" << std::endl << GFLOPS_S << "GFLOP/S" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}