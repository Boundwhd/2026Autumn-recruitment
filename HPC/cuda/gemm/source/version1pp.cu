#include "hip/hip_runtime.h"
#include "gemm.cuh"

template<const int block_size>
void __global__ gemm_v1pp(const float* A, const float* B, float* C, int M, int K, int N){
    int row = (blockIdx.x * block_size) + (threadIdx.x / block_size);
    int col = (blockIdx.y * block_size) + (threadIdx.x % block_size);

    if (row < M && col < N) {
        float sum = 0.0;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gemm_version1pp(const float* A, const float* B, float* C, int M, int K, int N) {
    dim3 block_size(32 * 32);
    dim3 grid_size(((M + 32 - 1) / 32), ((N + 32 - 1) / 32));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gemm_v1pp<32><<<grid_size, block_size>>>(A, B, C, M, K, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float GFLOPS_S = (2.0 * K * N * M) / milliseconds / 1e6;

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v1pp: " << milliseconds << " ms" << std::endl << GFLOPS_S << "GFLOP/S" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}