#include "hip/hip_runtime.h"
#include "gemm.cuh"
template<const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void gemm_v5(float* A, float* B, float* C, int M, int K, int N) {
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;
  
    const int threadCol = threadIdx.x % (BN / TN);
    const int threadRow = threadIdx.x / (BN / TN);
  
    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];
  
    A += cRow * BM * K;
    B += cCol * BN;
    C += cRow * BM * N + cCol * BN;
  
    const uint innerRowA = threadIdx.x / (BK / 4);
    const uint innerColA = threadIdx.x % (BK / 4);
    const uint innerRowB = threadIdx.x / (BN / 4);
    const uint innerColB = threadIdx.x % (BN / 4);

    float threadResults[TM * TN] = {0.0f};
    float regM[TM] = {0};
    float regN[TN] = {0};

    for (int bkIdx = 0; bkIdx < K; bkIdx += BK) {
        float4 tmp1 = reinterpret_cast<float4*>(&A[innerRowA * K + innerColA * 4])[0];
        float4 tmp2 = reinterpret_cast<float4*>(&A[(innerRowA + BM/2) * K + innerColA * 4])[0];
        
        As[(innerColA * 4 + 0) * BM + innerRowA] = tmp1.x;
        As[(innerColA * 4 + 1) * BM + innerRowA] = tmp1.y;
        As[(innerColA * 4 + 2) * BM + innerRowA] = tmp1.z;
        As[(innerColA * 4 + 3) * BM + innerRowA] = tmp1.w;

        As[(innerColA * 4 + 0) * BM + innerRowA + BM/2] = tmp2.x;
        As[(innerColA * 4 + 1) * BM + innerRowA + BM/2] = tmp2.y;
        As[(innerColA * 4 + 2) * BM + innerRowA + BM/2] = tmp2.z;
        As[(innerColA * 4 + 3) * BM + innerRowA + BM/2] = tmp2.w;

        reinterpret_cast<float4*>(&Bs[innerRowB * BN + innerColB * 4])[0] = 
            reinterpret_cast<float4*>(&B[innerRowB * N + innerColB * 4])[0];

        reinterpret_cast<float4*>(&Bs[(innerRowB + BK/2) * BN + innerColB * 4])[0] =
            reinterpret_cast<float4*>(&B[(innerRowB + BK/2) * N + innerColB * 4])[0];

        __syncthreads();

        A += BK;
        B += BK * N;

        for (uint dotIdx = 0; dotIdx < BK; dotIdx++) {
            for (int i = 0; i < TM; i++) {
                regM[i] = As[dotIdx * BM + (TM * threadRow + i)];
            }
            for (int j = 0; j < TN; j++) {
                regN[j] = Bs[dotIdx * BN + (TN * threadCol + j)];
            }
            for (int i = 0; i < TM; i++) {
                for (int j = 0; j < TN; j++) {
                    threadResults[i * TN + j] += regM[i] * regN[j];
                }
            }
        }
        __syncthreads();
    }

    for (int i = 0; i < TM; i++) {
        for (int j = 0; j < TN; j += 4) {
            float4 tmp = reinterpret_cast<float4*>(&C[(threadRow * TM + i) * N  + threadCol * TN + j])[0];
            tmp.x = threadResults[i * TN + j];
            tmp.y = threadResults[i * TN + j + 1];
            tmp.z = threadResults[i * TN + j + 2];
            tmp.w = threadResults[i * TN + j + 3];
            reinterpret_cast<float4 *>(&C[(threadRow * TM + i) * N + threadCol * TN + j])[0] = tmp;
        }
    }
}


void gemm_version5(float* A, float* B, float* C, int M, int K, int N) {
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    dim3 gridDim(DIV_CEIL(N, BN), DIV_CEIL(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gemm_v5<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(A, B, C, M ,K ,N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float GFLOPS_S = (2.0 * K * N * M) / milliseconds / 1e6;

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v5: " << milliseconds << " ms" << std::endl << GFLOPS_S << "GFLOP/S" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}