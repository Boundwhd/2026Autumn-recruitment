#include "hip/hip_runtime.h"
#include "gemm.cuh"

template <const int BM, const int BN, const int BK, const int TM>
void __global__ gemm_v3(const float* A, const float* B, float* C, int M, int K, int N) {
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;

    int threadRow = threadIdx.x / BN;
    int threadCol = threadIdx.x % BN;

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    A += cRow * K * BM;
    B += cCol * BN;
    C += cRow * K * BM + cCol * BN;


    const uint innerRowA = threadIdx.x / BK;
    const uint innerColA = threadIdx.x % BK;
    const uint innerRowB = threadIdx.x / BN;
    const uint innerColB = threadIdx.x % BN;

    float threadResult[TM] = {0.0};

    for (int bkidx = 0; bkidx < K; bkidx += BK) {
        As[innerRowA * BK + innerColA] = A[innerRowA * K + innerColA];
        Bs[innerRowB * BN + innerColB] = B[innerRowB * N + innerColB];

        __syncthreads();

        A += BK;
        B += N * BK;

        for (int i = 0; i < BK; i++) {
            float tmp_B = Bs[i * BN + threadCol];
            for (int j = 0; j < TM; j++) {
                threadResult[j] += As[(threadRow * TM + j) * BK + i] * tmp_B;
            }
        }
        __syncthreads();
    }

    for (int i = 0; i < TM; i++) {
        C[(threadRow * TM + i) * N + threadCol] = threadResult[i]; 
    }
}

void gemm_version3(const float* A, const float* B, float* C, int M, int K, int N) {
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    dim3 gridDim(((N + BN - 1) / BN), ((M + BM - 1) / BM));
    dim3 blockDim((BM * BN) / TM);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemm_v3<BM, BN, BK, TM><<<gridDim, blockDim>>>(A, B, C, M ,K ,N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v3: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}