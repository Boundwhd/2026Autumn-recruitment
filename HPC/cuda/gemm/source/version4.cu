#include "hip/hip_runtime.h"
#include "gemm.cuh"

template<const int BM, const int BN, const int BK, const int TM, const int TN>
void __global__ gemm_v4(const float* A, const float* B, float* C, int M, int K, int N) {
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;

    const uint totalResultsBlocktile = BM * BN;
    const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TM);

    const int threadCol = threadIdx.x % (BN / TN);
    const int threadRow = threadIdx.x % (BN / TN);

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    A += cRow * K * BM;
    B += cCol * BK;
    C += cRow * K * BN + cCol * BK;

    const uint innerRowA = threadIdx.x / BK;
    const uint innerColA = threadIdx.x % BK;
    const uint strideA = numThreadsBlocktile / BK;

    const uint innerRowB = threadIdx.x / BN;
    const uint innerColB = threadIdx.x % BN;
    const uint strideB = numThreadsBlocktile / BN;

    float threadResults[TM * TN] = {0.0};
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    for (int bkidx = 0; bkidx < K; bkidx += BK) {

        for (uint loadOffset = 0; loadOffset < BM; loadOffset += strideA) {
            As[(innerRowA + loadOffset) * BK + innerColA] =
                A[(innerRowA + loadOffset) * K + innerColA];
        }

        for (uint loadOffset = 0; loadOffset < BK; loadOffset += strideB) {
            Bs[(innerRowB + loadOffset) * BN + innerColB] =
                B[(innerRowB + loadOffset) * N + innerColB];
        }
        __syncthreads();

        A += BK;
        B += BK * N;

        for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
            for (int i = 0; i < TM; i++) {
                regM[i] = As[(threadRow * TM + i) * BK + dotIdx];
            }
            for (int i = 0; i < TN; i++) {
                regN[i] = Bs[(dotIdx * BN) + (threadCol * TN) + i];
            }

            for (int i = 0; i < TM; i++) {
                for (int j = 0; j < TN; j++) {
                    threadResults[i * TN + j] += regM[i] * regN[j];
                }
            }
        }
        __syncthreads();
    }

    for (int i = 0; i < TM; i++) {
        for (int j = 0; j < TN; j++) {
            C[(threadRow * TM + i) * N + threadCol * TM + j] = threadResults[i * TN + j];
        }
    }
}


void gemm_version4(const float* A, const float* B, float* C, int M, int K, int N) {
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    dim3 gridDim(DIV_CEIL(N, BN), DIV_CEIL(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gemm_v4<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(A, B, C, M ,K ,N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float GFLOPS_S = 4 / milliseconds / (0.001f);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v4: " << milliseconds << " ms" << std::endl << GFLOPS_S << "GFLOP/S" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}