#include "hip/hip_runtime.h"
#include "gemm.cuh"

template<const int BLOCK_SIZE>
void __global__ gemm_v2(const float* A, const float* B, float* C, int M, int K, int N) {
    int crow = blockIdx.y;
    int ccol = blockIdx.x;

    int row_thread = threadIdx.x / BLOCK_SIZE;
    int col_thread = threadIdx.x % BLOCK_SIZE;

    A += crow * BLOCK_SIZE * K;
    B += ccol * BLOCK_SIZE;
    C += crow * BLOCK_SIZE * N + ccol * BLOCK_SIZE;

    __shared__ float AS[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float BS[BLOCK_SIZE * BLOCK_SIZE];

    float sum = 0.0;
    for (int bkidx = 0; bkidx < K; bkidx += BLOCK_SIZE) {
        AS[row_thread * BLOCK_SIZE + col_thread] = A[row_thread * K + col_thread];
        BS[row_thread * BLOCK_SIZE + col_thread] = B[row_thread * N + col_thread];

        __syncthreads();

        A += BLOCK_SIZE;
        B += BLOCK_SIZE * N;

        for (int i = 0; i < BLOCK_SIZE; i++) {
            sum += AS[row_thread * BLOCK_SIZE + i] * BS[i * BLOCK_SIZE + col_thread];
        }
        
        __syncthreads();
    }

    C[row_thread * N + col_thread] = sum;
}


void gemm_version2(const float* A, const float* B, float* C, int M, int K, int N) {
    const int BLOCK_SIZE = 32;
    int block_size = 1024;
    dim3 grid_size((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemm_v2<BLOCK_SIZE><<<grid_size, block_size>>>(A, B, C, M, K, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v2: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}