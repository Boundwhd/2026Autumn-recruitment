#include "hip/hip_runtime.h"
#include "gemm.cuh"

void __global__ gemm_v1(const float* A, const float* B, float* C, int M, int K, int N){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gemm_version1(const float* A, const float* B, float* C, int M, int K, int N) {
    dim3 block_size(16, 16);
    dim3 grid_size(((N + 16 - 1) / 16), ((M + 16 - 1) / 16));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gemm_v1<<<grid_size, block_size>>>(A, B, C, M, K, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float GFLOPS_S = 4 / milliseconds / (0.001f);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v1: " << milliseconds << " ms" << std::endl << GFLOPS_S << "GFLOP/S" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}