#include "hip/hip_runtime.h"
#include "gemm.cuh"

void __global__ gemm_v1(const float* A, const float* B, float* C, int M, int K, int N){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void gemm_version1(const float* A, const float* B, float* C, int M, int K, int N) {
    dim3 block_size(32, 32);
    dim3 grid_size(((N + 32 - 1) / 32), ((M + 32 - 1) / 32));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemm_v1<<<grid_size, block_size>>>(A, B, C, M, K, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemm_v1: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}