#include <hipblas.h>
#include <gemv.cuh>

void gemv_cublas(const float* A, const float* B, float* C, int M, int N) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, B, M, A, 1, &beta, C, 1);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemv_cublas: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipblasDestroy(handle);
}