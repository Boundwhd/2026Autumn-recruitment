#include "hip/hip_runtime.h"
#include "gemv.cuh"
/**
 * 采取float4向量化指令，减少指令数据
 * 在小数据规模上无太大提升
 * 优化的是Icache，提高其访存命中
 */
void __global__ gemv_v2(const float* A, const float* B, float* C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    float4* A_4 = reinterpret_cast<float4*>(const_cast<float*>(A));
    float4* B_4 = reinterpret_cast<float4*>(const_cast<float*>(B));
    if (row < N) {
        float sum = 0;
        int M_4 = M / 4;
        for (int i = 0; i < M / 4; i++) {
            sum += A_4[i].x * B_4[row * M_4 + i].x;
            sum += A_4[i].y * B_4[row * M_4 + i].y;
            sum += A_4[i].z * B_4[row * M_4 + i].z;
            sum += A_4[i].w * B_4[row * M_4 + i].w;
        }
        
        for (int i = M_4 * 4; i < M; i++) {
            sum += A[i] * B[row * M + i];
        }

        C[row] = sum;
    }
}

void gemv_version2(const float* A, const float* B, float* C, int M, int N) {
    int block_size = 32;
    int grid_size = (N + block_size - 1) / block_size;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemv_v2<<<grid_size, block_size>>>(A, B, C, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app);
    if (outfile.is_open()) {
        outfile << "gemv_v2: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
}