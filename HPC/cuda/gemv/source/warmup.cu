#include "hip/hip_runtime.h"
#include "gemv.cuh"
__global__ void warmup_kernel() {

}

void warm_up() {
    warmup_kernel<<<32, 32>>>();
    hipDeviceSynchronize(); 

    int *d_temp;
    hipMalloc((void**)&d_temp, sizeof(int));
    hipFree(d_temp);
}