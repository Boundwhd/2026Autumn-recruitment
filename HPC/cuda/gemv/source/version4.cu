#include "hip/hip_runtime.h"
#include "gemv.cuh"

void __global__ gemv_v4(const float* A, const float* B, float* C, int M, int N) {
    int warp_id = threadIdx.x / warpSize;
    int lane_id = threadIdx.x % warpSize;
    int row = (blockDim.x / warpSize) * blockIdx.x + warp_id;

    if (row < N) {
        float sum = 0;
        for (int i = lane_id; i < M; i += warpSize) {
            sum += A[i] * B[row * N + i];
        }

        for (int offset = 16; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }

        if (lane_id == 0) {
            C[row] = sum;
        }
    }
}

void gemv_version4(const float* A, const float* B, float* C, int M, int N) {
    int block_size = 1024;
    int grid_size = N;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
        
    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemv_v4<<<grid_size, block_size>>>(A, B, C, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemv_v4: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}