#include "hip/hip_runtime.h"
#include "gemv.cuh"

/**
 * naive实现
 * 一个thread负责一个输出的计算
 */

void __global__ gemv_v1(const float* A, const float* B, float* C, int M, int N) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < N) {
        float sum = 0;
        for (int i = 0; i < M; i++) {
            sum += A[i] * B[row * M + i];
        }
        C[row] = sum;
    }
}

void gemv_version1(const float* A, const float* B, float* C, int M, int N) {
    int block_size = 8;
    int grid_size = (N + block_size - 1) / block_size;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
        
    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemv_v1<<<grid_size, block_size>>>(A, B, C, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemv_v1: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
