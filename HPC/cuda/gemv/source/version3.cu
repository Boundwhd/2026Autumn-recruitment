#include "hip/hip_runtime.h"
#include "gemv.cuh"

/**
 * 一个输出使用一个block计算
 * 每个block配置1024线程
 */


void __global__ gemv_v3(const float* A, const float* B, float* C, int M, int N) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    __shared__ float mem[32];

    float sum = 0;
    for (int i = col; i < N; i += blockDim.x) {
        sum += A[i] * B[row * M + i];
    }

    for (int offset = 16; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0XFFFFFFFF, sum, offset);
    }

    __syncthreads();

    int warptid = col % warpSize;
    int warpid = col / warpSize;
    if (warptid == 0) {
        mem[warpid] = warpid < (N / warpSize) ? sum : 0;
    }

    __syncthreads();

    if (warpid == 0) {
        sum = mem[warpid];
    }

    __syncthreads();

    for (int offset = 16; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0XFFFFFFFF, sum, offset);
    }

    if (warpid == 0 && warpid == 0) {
        C[row] = sum;
    }
}


void gemv_version3(const float* A, const float* B, float* C, int M, int N) {
    int block_size = 1024;
    int grid_size = N;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
        
    hipEventRecord(start);
    for (int i = 0; i < COUNT; i++) {
        gemv_v3<<<grid_size, block_size>>>(A, B, C, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::ofstream outfile("kernel_timings.txt", std::ios::app); 
    if (outfile.is_open()) {
        outfile << "gemv_v3: " << milliseconds << " ms" << std::endl << std::endl;
        outfile.close();
    } else {
        std::cerr << "Failed to open file for writing!" << std::endl;
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}